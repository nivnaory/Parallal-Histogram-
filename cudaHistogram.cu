#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "myProto.h"

__global__ void HistogramKernel(const int* a , int* dev_globalHist, int size)
{
    //Each block has a private histogram that can be accessed through the shared memory between all the threads in the block.
    __shared__ int sharedHistogram[NUMBERS + 1];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = threadIdx.x + 1;
    sharedHistogram[threadIdx.x] = 0;
    if(i < size){
    	atomicAdd(&sharedHistogram[a[i]], 1);
    }
    __syncthreads();
    atomicAdd(&dev_globalHist[j],sharedHistogram[j]);
}

int HistogramWithCuda(int rank,int* arr, int size, int* totalEachProcHistogram)
{
    int* dev_arr = 0;
    int globalHist[NUMBERS + 1] = {0};
    int *dev_globalHist;
    hipError_t cudaStatus;   
  

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    // Allocate GPU buffers for data (input).
    cudaStatus = hipMalloc((void**)&dev_arr, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    // Allocate GPU buffers for gloabl histogram.
    cudaStatus = hipMalloc((void**)&dev_globalHist, (NUMBERS+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }


    // Copy input data from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 1;
    }

    cudaStatus = hipMemcpy(dev_globalHist,globalHist ,(NUMBERS +1) * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return 1;
        }

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(size + threadsPerBlock - 1) / threadsPerBlock;
    HistogramKernel <<<blocksPerGrid, threadsPerBlock >>> (dev_arr,dev_globalHist , size);

    // Copy output global histogram from GPU buffer to host memory.
    cudaStatus = hipMemcpy(globalHist, dev_globalHist, (NUMBERS+1) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return 1;
    }

     for (int i=0;i<=NUMBERS;i++)
     {
       totalEachProcHistogram[i]+=globalHist[i];
     }
     
    hipFree(dev_arr);
    hipFree(dev_globalHist);

    return 0;
}

